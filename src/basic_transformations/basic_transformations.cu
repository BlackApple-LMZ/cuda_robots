#include "hip/hip_runtime.h"
#include "basic_transformations/basic_transformations.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/copy.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/extrema.h>


__global__ void kernel_cudaWarmUpGPU()
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	ind = ind + 1;
    //printf("Device Name : %d.\n", ind); //每个warp按照顺序打印 512 对应了16个warp 这16个warp打印顺序是不确定的
}

hipError_t cudaWarmUpGPU()
{
	kernel_cudaWarmUpGPU<<<1,1>>>();
	hipDeviceSynchronize(); //Wait for compute device to finish 等所有线程都运行结束
	return hipGetLastError(); 
}

__global__ void kernel_cudaTransformPoints(pcl::PointXYZ *d_point_cloud, int number_of_points, float *d_matrix)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;




	//因为分配的线程数量大于实际的点云数量 这里的判断是防止溢出
	if(ind<number_of_points)
	{
		float vSrcVector[3] = {d_point_cloud[ind].x, d_point_cloud[ind].y, d_point_cloud[ind].z};
		float vOut[3];

		//对每个点计算转换后的坐标
		vOut[0]=d_matrix[0]*vSrcVector[0]+d_matrix[4]*vSrcVector[1]+d_matrix[8]*vSrcVector[2]+d_matrix[12];
   	 	vOut[1]=d_matrix[1]*vSrcVector[0]+d_matrix[5]*vSrcVector[1]+d_matrix[9]*vSrcVector[2]+d_matrix[13];
    	vOut[2]=d_matrix[2]*vSrcVector[0]+d_matrix[6]*vSrcVector[1]+d_matrix[10]*vSrcVector[2]+d_matrix[14];

		d_point_cloud[ind].x = vOut[0];
		d_point_cloud[ind].y = vOut[1];
		d_point_cloud[ind].z = vOut[2];
	}
}

hipError_t cudaTransformPoints(int threads, pcl::PointXYZ *d_point_cloud, int number_of_points, float *d_matrix)
{
    //分配足够的threads 计算点云变换
    //kernel前两个参数的类型 决定了 grid 和 block的维度 这里都是1维的 也可以是dim3 维的
	kernel_cudaTransformPoints<<<number_of_points/threads+1,threads>>>
		(d_point_cloud, number_of_points, d_matrix);

	hipDeviceSynchronize();
	return hipGetLastError();
}

__global__ void kernel_cudaRemovePointsInsideSphere
	(pcl::PointXYZ *d_point_cloud, bool *d_markers, int number_of_points, float sphere_radius)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;

	if(ind<number_of_points)
	{
		float x = d_point_cloud[ind].x;
		float y = d_point_cloud[ind].y;
		float z = d_point_cloud[ind].z;
	
		float distance = (x*x + y*y + z*z);
	
		if(distance < sphere_radius * sphere_radius)
		{
			d_markers[ind] = false;
		}else
		{
			d_markers[ind] = true;
		}
	}
}

hipError_t cudaRemovePointsInsideSphere(int threads, pcl::PointXYZ *d_point_cloud, 
		bool *d_markers, int number_of_points, float sphere_radius)
{
    //    dim3 dimGrid(number_of_points/threads+1,1,1);
    //    dim3 dimBlock(threads,1,1);
	kernel_cudaRemovePointsInsideSphere<<<number_of_points/threads+1,threads>>>
		(d_point_cloud,	d_markers, number_of_points, sphere_radius);
		
	hipDeviceSynchronize();
	return hipGetLastError();
}






